
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void){
    printf("hello world from gpu!\n");
}

int main(void){
    // hello from cpu
    printf("hello from cpu!\n");
    // 1 thread block and 10 threads
    helloFromGPU <<<1,10>>>();
    hipDeviceReset();
    return 0;
}